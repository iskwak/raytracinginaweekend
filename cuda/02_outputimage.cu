// Chapter 2: Creating a ppm

#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if(result) {
        std::cerr << "CUDAS error = " << static_cast<unsigned int>(result) << " at " <<file  << ": " << line << " '" << func << "'\n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}


int main() {
    int nx = 256;
    int ny = 256;
    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    float *fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    int tx = 8;
    int ty = 8;

    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    std::cout <<"P3\n" << nx << " " << ny << "\n255\n";
    for(int j = ny - 1; j >= 0; --j) {
        std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
        for(int i = 0; i < nx; ++i) {
            size_t pixel_index = j * 3 * nx + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];

            int ir = static_cast<int>(255.999 * r);
            int ig = static_cast<int>(255.999 * g);
            int ib = static_cast<int>(255.999 * b);

            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    std::cerr << "\nDone.\n";

    checkCudaErrors(hipFree(fb));
    return 0;
}

